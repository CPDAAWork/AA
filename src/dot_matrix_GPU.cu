#include "hip/hip_runtime.h"
/**************************************************************
 *
 * --== Simple CUDA kernel ==--
 * author: ampereira
 *
 *
 * Fill the rest of the code
 *
 * Insert the functions for time measurement in the correct
 * sections (i.e. do not account for filling the vectors with random data)
 *
 * Before compile choose the CPU/CUDA version by running the bash command:
 *     export CUDA=yes    or    export CUDA=no
 *
 * The stencil array size must be set to the SIZE #define, i.e., float stencil[SIZE];
 **************************************************************/
#include <stdio.h>
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <sys/time.h>
#include "more/matrix.h"

#define TIME_RESOLUTION 1000000	// time measuring resolution (us)

#define SIZE 1024
#define SIZE_MATRIX SIZE*SIZE
#define NUM_BLOCKS SIZE
#define NUM_THREADS_PER_BLOCK SIZE
#define NUM_THREADS NUM_BLOCKS*NUM_THREADS_PER_BLOCK

void printfMatrix(float *c){
	for(int i=0; i<SIZE; i++){
		for(int j=0; j<SIZE; j++){
			printf("c[i][j] = %f; ", c[i*SIZE+j]);
		}
		printf("\n");
	}
}

using namespace std;

long long unsigned cpu_time;
hipEvent_t start, stop;
struct timeval t;

void startTime (void) {
	gettimeofday(&t, NULL);
	cpu_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;
}

void stopTime (void) {
	gettimeofday(&t, NULL);
	long long unsigned final_time = t.tv_sec * TIME_RESOLUTION + t.tv_usec;

	final_time -= cpu_time;

	cout << final_time << " us have elapsed" << endl;
}

// These are specific to measure the execution of only the kernel execution - might be useful
void startKernelTime (void) {
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
}

void stopKernelTime (void) {
	hipEventRecord(stop);

	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);

	cout << milliseconds << " ms have elapsed for the kernel execution" << endl;
}

void checkCUDAError (const char *msg) {
	hipError_t err = hipGetLastError();
	if( hipSuccess != err) {
		cerr << "Cuda error: " << msg << ", " << hipGetErrorString(err) << endl;
		exit(-1);
	}
}

// Fill the input parameters and kernel qualifier
__global__ void stencilKernel (const float *_a,const float *_b, float *_c) {
	int i = blockIdx.x*SIZE;
	int j = threadIdx.x;
	int j_b = threadIdx.x*SIZE;

	float temp=0;
	__shared__ float As[SIZE];

    As[j] = _a[i+j];

    __syncthreads();

	for (int k=0;k<SIZE;++k)
		temp+=As[k]*_b[j_b+k];

	_c[i+j] = temp;
}

// Fill the input parameters and kernel qualifier
__global__ void teste (float *_a, float *_b, float *_c) {
	int i = blockIdx.x*SIZE;
	int j = threadIdx.x;
	_c[i+j] = i+j;
}

// Fill with the code required for the GPU stencil (mem allocation, transfers, kernel launch of stencilKernel)
void stencilGPU (void) {
	// Size of the array
	int bytes = SIZE_MATRIX*(sizeof(float));

	printf("Bytes: %d\n",bytes);

	// pointers to the device
	float *a_device,*b_device,*c_device;
	float *a,*b,*c;

	a = (float*)malloc(bytes);
	b = (float*)malloc(bytes);
	c = (float*)malloc(bytes);

	// allocate the memory on the device
	hipMalloc(&a_device, bytes);
	hipMalloc(&b_device, bytes);
	hipMalloc(&c_device, bytes);

	checkCUDAError("mem allocation\n");

	// fills the arrays
	for (int i = 0; i < SIZE; ++i) {
		for(int j = 0; j < SIZE; ++j){
			a[i*SIZE+j] = 1;
			b[i*SIZE+j] = i+1;
			c[i*SIZE+j] = 0;
		}
	}

	// copy inputs to the device
	hipMemcpy(a_device, a, bytes, hipMemcpyHostToDevice);
	hipMemcpy(b_device, b, bytes, hipMemcpyHostToDevice);
	hipMemcpy(c_device, c, bytes, hipMemcpyHostToDevice);

	checkCUDAError("memcpy h->d");
	// declare the number of blocks per grid and the number of threads per block
    // use 1 to 512 threads per block

	startTime();
	stencilKernel <<<NUM_BLOCKS,NUM_THREADS_PER_BLOCK>>> (a_device, b_device, c_device);
	stopTime();

	checkCUDAError("kernel invocation");
	// copy the output to the host
	hipMemcpy(c, c_device,bytes, hipMemcpyDeviceToHost);
	checkCUDAError("memcpy d->h");

	//printfMatrix(c);

	// free the device memory
	hipFree(a_device);hipFree(b_device);hipFree(c_device);
	free(a);free(b);free(c);

	checkCUDAError("mem free");
}

// Fill with the code required for the CPU stencil
int main (int argc, char** argv) {

	printf("NUM_THREADS %d\n", NUM_THREADS_PER_BLOCK);
	
	printf("GPU ");
	startTime();
	stencilGPU();
	stopTime();

	return 0;
}
